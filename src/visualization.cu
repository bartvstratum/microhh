#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2020 Chiel van Heerwaarden
 * Copyright (c) 2011-2020 Thijs Heus
 * Copyright (c) 2014-2020 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <iostream>

#include "master.h"
#include "grid.h"
#include "fields.h"
#include "timeloop.h"
#include "visualization.h"

#include <GL/glew.h>
#include <GL/freeglut.h>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

namespace
{
    __device__
    unsigned char clip(int n)
    {
      return n > 255 ? 255 : (n < 0 ? 0 : n);
    }

    template<typename TF> __global__
    void set_values(
            uchar4* image,
            const TF* const __restrict__ fld,
            const int width, const int height,
            const TF vmin, const TF vmax,
            const int j_index,
            const int istart, const int iend,
            const int kstart, const int kend,
            const int icells, const int ijcells)
    {
        // Column/row index
        const int i = blockIdx.x*blockDim.x + threadIdx.x;
        const int k = blockIdx.y*blockDim.y + threadIdx.y;

        if ( (i >= width) || (k >= height) )
            return;

        const int ij  = i+(height-k-1)*width;
        const int ijk = (i+istart) + j_index*icells + (k+kstart)*ijcells;

        const TF rel_val = (fld[ijk] - vmin) / (vmax - vmin);
        const unsigned char rel_val_i = clip(rel_val * 255);

        image[ij].x = rel_val_i;
        image[ij].y = rel_val_i;
        image[ij].z = rel_val_i;
        image[ij].w = 255;
    }
}

template <typename TF>
void Visualization<TF>::exec(Timeloop<TF>& timeloop)
{
    if (timeloop.in_substep())
        return;

    auto& gd = grid.get_grid_data();

    // This only works with freeglut, but allows us
    // to use OpenGL without the glutMainLoop()
    glutMainLoopEvent();

    // Map graphics resources so that CUDA can access it
    uchar4 *d_out = 0;
    hipGraphicsMapResources(1, &cuda_pbo_resource, 0);
    hipGraphicsResourceGetMappedPointer(
            (void **)&d_out, NULL, cuda_pbo_resource);

    // Launch kernel
    const int blocki = 32;
    const int blockj = 32;
    const int gridi  = gd.imax/blocki + (gd.imax%blocki > 0);
    const int gridj  = gd.kmax/blockj + (gd.kmax%blockj > 0);

    dim3 grid_gpu(gridi, gridj, 1);
    dim3 block_gpu(blocki, blockj, 1);

    const TF vmin = 300;
    const TF vmax = 303;

    set_values<TF><<<grid_gpu, block_gpu>>>(
            d_out, fields.ap.at("th")->fld_g,
            width, height, vmin, vmax, gd.jstart,
            gd.istart, gd.iend,
            gd.kstart, gd.kend,
            gd.icells, gd.ijcells);

    // Unmap graphics resources
    hipGraphicsUnmapResources(1, &cuda_pbo_resource, 0);

    // Draw texture
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, width, height, 0, GL_RGBA, GL_UNSIGNED_BYTE, NULL);
    glEnable(GL_TEXTURE_2D);
    glBegin(GL_QUADS);
    glTexCoord2f(0.0f, 0.0f); glVertex2f(0, 0);
    glTexCoord2f(0.0f, 1.0f); glVertex2f(0, height);
    glTexCoord2f(1.0f, 1.0f); glVertex2f(width, height);
    glTexCoord2f(1.0f, 0.0f); glVertex2f(width, 0);
    glEnd();
    glDisable(GL_TEXTURE_2D);

    // Swap buffer to show updated image
    glutSwapBuffers();
}

template class Visualization<double>;
template class Visualization<float>;
